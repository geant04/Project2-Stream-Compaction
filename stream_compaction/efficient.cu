#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#include <iostream>

// Debugging defines, toggling off/on chunks of code helps me figure stuff out
#define SIMPLE_EFFICIENT_SCAN 1
#define ENABLE_DOWNSWEEP 1

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upsweep(int n, int stride, int* dev_odata)
        {
            int index = blockDim.x * blockIdx.x + threadIdx.x;

            if (index >= n || index >= (n / stride / 2))
            {
                return;
            }

            // stride = 2^d, d \in [0, log2(n) - 1]
            int strideMult2 = stride * 2;
            if (index < (n / strideMult2))
            {
                if ((n/strideMult2) == 1)
                {
                    // Last element clear at final stage
                    dev_odata[index + strideMult2 - 1] = 0;
                    return;
                }

                int writeIndex = index * strideMult2;
                dev_odata[writeIndex + strideMult2 - 1] += dev_odata[writeIndex + stride - 1];
            }
        }

        __global__ void downsweep(int n, int stride, int* dev_odata)
        {
            int index = blockDim.x * blockIdx.x + threadIdx.x;

            if (index >= n || index >= (n / stride))
            {
                return;
            }

            int strideDiv2 = stride / 2;

            if (index < (n / stride))
            {
                int writeIndex = index * stride;

                int left = dev_odata[writeIndex + strideDiv2 - 1];
                int right = dev_odata[writeIndex + stride - 1];

                dev_odata[writeIndex + strideDiv2 - 1] = right;
                dev_odata[writeIndex + stride - 1] += left;
            }
        }

        // Shared mem implementation
        __global__ void optimizedSharedScan(int n, int* dev_odata)
        {
            extern __shared__ float sharedData[];
            
            int threadID = threadIdx.x;

            // This only works for thread up to max block size for now.
            // I'll have to spend a few hours to figure out how to fit everything on a SM.
            // There are also some potential bank conflicts.
            sharedData[threadID] = dev_odata[threadID];

            __syncthreads();

            // Shared mem upsweep, nothing too different.
            for (int d = n/2; d > 0; d >>= 1)
            {
                if (threadID < d)
                {
                    int strideMult2 = n/d;
                    int stride = strideMult2 >> 1;

                    int writeIndex = threadID * strideMult2;
                    sharedData[writeIndex + strideMult2 - 1] += sharedData[writeIndex + stride - 1];
                }

                __syncthreads();
            }

            // clear last element... this will waste a few cycles
            if (threadID < 1)
            {
                sharedData[n - 1] = 0;
            }

            __syncthreads();

#if ENABLE_DOWNSWEEP
            for (int d = 1; d < n; d <<= 1)
            {
                if (threadID < d)
                {
                    int stride = n/d;
                    int strideDiv2 = stride >> 1;

                    int writeIndex = threadID * stride;

                    int left = sharedData[writeIndex + strideDiv2 - 1];
                    int right = sharedData[writeIndex + stride - 1];

                    sharedData[writeIndex + strideDiv2 - 1] = right;
                    sharedData[writeIndex + stride - 1] += left;
                }

                __syncthreads();
            }

            __syncthreads();
#endif


            dev_odata[threadID] = sharedData[threadID];
        }


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

        void scanDispatch(int blocks, int blockSize, int n, int stages, int &stride, int *dev_odata)
        {
            // upsweep, write to temp buffer
            for (int d = 0; d <= stages; d++)
            {
                upsweep<<<blocks, blockSize>>>(n, stride, dev_odata);
                stride <<= 1;
            }

            for (int d = 0; d <= stages; d++)
            {
                downsweep<<<blocks, blockSize>>>(n, stride, dev_odata);
                stride >>= 1;
            }
        }


        void scan(int n, int *odata, const int *idata) {
            int paddedN = 1 << ilog2ceil(n);

            int *dev_odata;

            int sizeOfData = paddedN * sizeof(int);

            hipMalloc((void**)&dev_odata, sizeOfData);

            // Copy idata to dev_odata first, this way we can easily modify in place
            hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            int blocks = (paddedN + blockSize - 1) / blockSize;

            int stages = ilog2(paddedN) - 1;
            int stride = 1;

            timer().startGpuTimer();
            
            scanDispatch(blocks, blockSize, paddedN, stages, stride, dev_odata);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeOfData, hipMemcpyDeviceToHost);
            
            hipFree(dev_odata);
        }

        void optimizedScan(int n, int *odata, const int *idata)
        {
            int paddedN = 1 << ilog2ceil(n);

            int *dev_odata;
            int sizeOfData = paddedN * sizeof(int);

            hipMalloc((void**)&dev_odata, sizeOfData);

            // Copy idata to dev_odata first, this way we can easily modify in place
            hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 512;
            int blocks = (paddedN + blockSize - 1) / blockSize;

            int stages = ilog2(paddedN);
            int stride = 1;

            int optimizedBlockSize = blockSize;
            int optimizedBlocks = blocks;

            int threadsToRun, optimizedN;

            timer().startGpuTimer();

            for (int d = 0; d < stages; d++)
            {
                upsweep<<<optimizedBlocks, optimizedBlockSize>>>(paddedN, stride, dev_odata);
                stride <<= 1;

                // Threads to run are halved
                threadsToRun = paddedN >> (d + 1);
                optimizedN = threadsToRun;

                threadsToRun = (threadsToRun >= blockSize) ? blockSize : threadsToRun;
                threadsToRun = (threadsToRun <= 32) ? 32 : threadsToRun;

                optimizedBlockSize = threadsToRun;
                optimizedBlocks = (optimizedN + optimizedBlockSize - 1) / optimizedBlockSize;
            }

#if ENABLE_DOWNSWEEP
            for (int d = 0; d < stages; d++)
            {
                downsweep<<<optimizedBlocks, optimizedBlockSize>>>(paddedN, stride, dev_odata);
                stride >>= 1;

                // Since N is padded to the nearest power of 2, this logic to compute # of threads is fine
                threadsToRun = 1u << (d + 1);
                optimizedN = threadsToRun;

                threadsToRun = (threadsToRun <= 32) ? 32 : threadsToRun;
                threadsToRun = (threadsToRun >= blockSize) ? blockSize : threadsToRun;

                optimizedBlockSize = threadsToRun;
                optimizedBlocks = (optimizedN + optimizedBlockSize - 1) / optimizedBlockSize;
            }
#endif
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeOfData, hipMemcpyDeviceToHost);
            
            hipFree(dev_odata);
        }

        void optimizedMemScan(int n, int *odata, const int *idata)
        {
            int paddedN = 1 << ilog2ceil(n);

            int *dev_odata;

            int sizeOfData = paddedN * sizeof(int);

            hipMalloc((void**)&dev_odata, sizeOfData);

            // Copy idata to dev_odata first, this way we can easily modify in place
            hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            int blocks = (paddedN + blockSize - 1) / blockSize;

            timer().startGpuTimer();

            // Ideally, we use one kernel per thread to reduce overhead from running MULTIPLE kernel dispatches.
            // This also takes care of early terminating warps early on, as we don't run more than 1 dispatch.
            optimizedSharedScan<<<blocks, blockSize>>>(paddedN, dev_odata);
            
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeOfData, hipMemcpyDeviceToHost);
            
            hipFree(dev_odata);
        }


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */

        __global__ void kernMapToBoolean(int n, int *dev_bitmap)
        {
            int index = blockDim.x * blockIdx.x + threadIdx.x;

            if (index >= n)
            {
                return;
            }

            dev_bitmap[index] = (dev_bitmap[index] > 0) ? 1 : 0;
        }

        __global__ void kernScatter(int n, int *dev_bitmap, int *dev_odata, int *dev_idata)
        {
            int index = blockDim.x * blockIdx.x + threadIdx.x;

            if (index >= n)
            {
                return;
            }

            if (dev_bitmap[index] > 0)
            {
                int scatterIndex = dev_odata[index];
                dev_odata[scatterIndex] = dev_idata[index];
            }
        }

        int compact(int n, int *odata, const int *idata) 
        {
            int paddedN = 1 << ilog2ceil(n);
            int sizeOfData = paddedN * sizeof(int);

            int *dev_idata;
            int *dev_odata;
            int *dev_bitmap;

            hipMalloc((void**)&dev_idata, sizeOfData);
            hipMalloc((void**)&dev_odata, sizeOfData);
            hipMalloc((void**)&dev_bitmap, sizeOfData);

            // Copy idata to dev_odata first, this way we can easily modify in place
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_bitmap, idata, n * sizeof(int), hipMemcpyHostToDevice);

            int blockSize = 128;
            int blocks = (paddedN + blockSize - 1) / blockSize;

            int stages = ilog2(paddedN) - 1;
            int stride = 1;

            timer().startGpuTimer();

            // Write to dev_bitmap, input is idata memcpyed to dev_bitmap
            kernMapToBoolean<<<blocks, blockSize>>>(paddedN, dev_bitmap);

            // Copy dev_bitmap info to dev_odata, this is needed so we can run scan
            hipMemcpy(dev_odata, dev_bitmap, sizeOfData, hipMemcpyDeviceToDevice);

            // Scan writes output to dev_odata
            scanDispatch(blocks, blockSize, paddedN, stages, stride, dev_odata);

            // Write to dev_odata with inputs idata, bitmap, and scanOutput, which is dev_odata at this point.
            kernScatter<<<blocks, blockSize>>>(paddedN, dev_bitmap, dev_odata, dev_idata);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, sizeOfData, hipMemcpyDeviceToHost);
            hipFree(dev_odata);
            hipFree(dev_idata);
            hipFree(dev_bitmap);

            return odata[paddedN - 1];
        }
    }
}
